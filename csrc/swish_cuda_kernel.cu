#include "hip/hip_runtime.h"
#include <torch/types.h>
#include <torch/extension.h>
// #include <ATen/cuda/CUDAApplyUtils.cuh>

#include "CUDAApplyUtils.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// TORCH_CHECK replaces AT_CHECK in PyTorch 1,2, support 1.1 as well.
#ifndef TORCH_CHECK
#define TORCH_CHECK AT_CHECK
#endif

#ifndef __CUDACC_EXTENDED_LAMBDA__
#error "please compile with --expt-extended-lambda"
#endif


namespace kernel {
#include "swish.h"

using at::cuda::CUDA_tensor_apply2;
using at::cuda::CUDA_tensor_apply3;
using at::cuda::TensorArgType;

template <typename scalar_t>
void
swish_forward(
  torch::Tensor &output,
  const torch::Tensor &input
) {
  CUDA_tensor_apply2<scalar_t,scalar_t>(
    output, input,
    [=] __host__ __device__ (scalar_t &out, const scalar_t &inp) {
      swish_fwd_func(out, inp);
    },
    TensorArgType::ReadWrite, TensorArgType::ReadOnly
  );
}

template <typename scalar_t>
void
swish_backward(
  torch::Tensor &grad_inp,
  const torch::Tensor &input,
  const torch::Tensor &grad_out
) {
  CUDA_tensor_apply3<scalar_t,scalar_t,scalar_t>(
    grad_inp, input, grad_out,
    [=] __host__ __device__ (scalar_t &grad_inp, const scalar_t &inp, const scalar_t &grad_out) {
      swish_bwd_func(grad_inp, inp, grad_out);
    },
    TensorArgType::ReadWrite, TensorArgType::ReadOnly, TensorArgType::ReadOnly
  );
}

} // namespace kernel

template <typename scalar_t>
__device__ __forceinline__ scalar_t sigmoid(scalar_t z) {
  return 1.0 / (1.0 + exp(-z));
}

void
swish_forward_cuda(
    torch::Tensor &output, const torch::Tensor &input
) {
  auto in_arg  = torch::TensorArg(input,  "input",  0),
       out_arg = torch::TensorArg(output, "output", 1);
  torch::checkAllDefined("swish_forward_cuda", {in_arg, out_arg});
  torch::checkAllSameGPU("swish_forward_cuda", {in_arg, out_arg});
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(), "swish_forward_cuda", [&] {
      kernel::swish_forward<scalar_t>(output, input);
  });
}

void
swish_backward_cuda(
  torch::Tensor &grad_inp, const torch::Tensor &input, const torch::Tensor &grad_out
) {
  auto gi_arg = torch::TensorArg(grad_inp, "grad_inp", 0),
       in_arg = torch::TensorArg(input,    "input",    1),
       go_arg = torch::TensorArg(grad_out, "grad_out", 2);
  torch::checkAllDefined("swish_backward_cuda", {gi_arg, in_arg, go_arg});
  torch::checkAllSameGPU("swish_backward_cuda", {gi_arg, in_arg, go_arg});
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(grad_inp.scalar_type(), "swish_backward_cuda", [&] {
      kernel::swish_backward<scalar_t>(grad_inp, input, grad_out);
  });
}
